#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, nullptr);
  return tv.tv_sec + 1e-6 * tv.tv_usec;
}

constexpr int m = 2;
constexpr int block_size = 128;

struct Node {
  int lock;
  int sum;

  __device__ void inc() {
    /*
    while (atomicCAS(&lock, 0, 1))
      ;
    sum += 1;
    atomicExch(&lock, 0);
     */

    for (int i = 0; i < 32; i++) {
      if (i == threadIdx.x % 32) {
        while (atomicExch(&lock, 1) == 1)
          ;
        // printf("locked\n");
        atomicExch(&lock, 0);
      }
    }
  }
};

__global__ void inc(Node *nodes) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  int b = i % m;
  nodes[b].inc();

  /*
  int warp_id = threadIdx.x % 32;
  int b = warp_id;
  int done = 0;
  if (true) {
    auto mask = __activemask();
    // printf("mask %d\n", mask);
    while (!__all_sync(mask, done)) {
      for (int k = 0; k < 32; k++) {
        if (k == warp_id && !done) {
          int &lock = nodes[b].lock;
          if (atomicCAS(&lock, 0, 1) == 0) {
            nodes[b].sum += 1;
            done = true;
            atomicExch(&lock, 0);
          }
        }
      }
    }
  } else {
    for (int k = 0; k < 32; k++) {
      if (k == warp_id) {
        int &lock = nodes[b].lock;
        while (atomicCAS(&lock, 0, 1))
          ;
        nodes[b].sum += 1;
        done = true;
        atomicExch(&lock, 0);
      }
    }
  }
  */
}

void mutex() {
  Node *a;

  hipMallocManaged(&a, m * sizeof(Node));

  for (int i = 0; i < 20; i++) {
    hipDeviceSynchronize();
    auto t = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipDeviceSynchronize();
    inc<<<1, 4>>>((Node *)a);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "device  " << milliseconds << std::endl;
    int sum = 0;
    for (int j = 0; j < m; j++) {
      sum += a[j].sum;
    }
    printf("sum %d\n", sum);
  }
  std::cout << std::endl;
}

template <typename T>
__device__ bool unique_in_warp(T val) {
  auto mask = __activemask();

  auto warpId = threadIdx.x % warpSize;

  bool has_following_eqiv = 0;
  for (int i = 1; i < warpSize; i++) {
    auto cond = warpId + i < warpSize;
    bool same = (cond & (val == __shfl_down_sync(mask, val, i)));
    has_following_eqiv = has_following_eqiv || (cond && same);
  }

  return !has_following_eqiv;
}

__device__ int elect_leader(int mask) {
  return __ffs(mask) - 1;
}

__global__ void elect(long long *addr) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  auto warpId = threadIdx.x % warpSize;

  /*
  if (unique_in_warp(addr)) {
    printf("%lld\n", addr);
  }
  */

  auto mask = __activemask();

  int uniques = __ballot_sync(mask, unique_in_warp(addr[i]));
  while (uniques) {
    int leader = elect_leader(uniques);
    if (warpId == leader) {
      printf("leader %d val %lld\n", leader, addr[i]);
    }
    uniques ^= 1 << leader;
  }
}

void elect_diff() {
  long long *a;

  hipMallocManaged(&a, 32 * sizeof(long long));

  for (int i = 0; i < 32; i++) {
    a[i] = i % 5;
  }

  for (int i = 0; i < 20; i++) {
    hipDeviceSynchronize();
    auto t = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipDeviceSynchronize();
    elect<<<1, 32>>>(a);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "device  " << milliseconds << std::endl;
  }
  std::cout << std::endl;
}

int main() {
  elect_diff();
}
